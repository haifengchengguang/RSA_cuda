
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>

#define R_size 129
#define k 1024
#define n_size 128

__global__ void get_square(unsigned char a[], unsigned int accumulator[], unsigned int n);
void square(unsigned char *a, unsigned char *c, unsigned int size);
__global__ void get_products(unsigned char a[], unsigned char b[], unsigned int accumulator[], unsigned int n);
void barrett_reduction(unsigned char *buf_cpu, unsigned char *buf_gpu, unsigned char *gpu_r, unsigned char *gpu_n, unsigned char *reduction, unsigned char *temp_cpu, unsigned char *temp_gpu, unsigned char *shifted_cpu, unsigned char *shifted_gpu, unsigned char *xprime_cpu, unsigned char *xprime_gpu, unsigned char *result, unsigned char *tmp, unsigned int exp_size, unsigned int *kernel_buf, unsigned int *transfer, unsigned char *n);
void multiplication(unsigned char *a, unsigned char *b, unsigned char *c, unsigned int size); 
char checkbit(unsigned char *exponent, unsigned int index_of_bit); 
void subtraction(unsigned char *a, unsigned char *b, unsigned char *c, unsigned int size); 
void bit_shift(unsigned char *a, unsigned char *b, unsigned int shift, unsigned int size_of_a); 
void exponentiation(unsigned char *gpu_message, unsigned char *exponent, unsigned char *gpu_ciphertext, unsigned char *m0_copy, unsigned char *reduction, unsigned char *buf_cpu, unsigned char *buf_gpu, unsigned char *temp_cpu, unsigned char *temp_gpu, unsigned char *shifted_cpu, unsigned char *shifted_gpu, unsigned char *xprime_cpu, unsigned char *xprime_gpu, unsigned char *result, unsigned char *tmp, unsigned int exponent_size, unsigned char *gpu_r, unsigned char *gpu_n, unsigned int *kernel_buf, unsigned int *transfer, unsigned char *n);

int main(int argc, char *argv[]) {

unsigned char *n = (unsigned char *) calloc((4*n_size + n_size), sizeof(char));
//申请内存
    n[0] = 0xcf;
    n[1] = 0x82;
    n[2] = 0x69;
    n[3] = 0x57;
    n[4] = 0x4d;
    n[5] = 0xe7;
    n[6] = 0x82;
    n[7] = 0x1a;
    n[8] = 0xe4;
    n[9] = 0x20;
    n[10] = 0x14;
    n[11] = 0x47;
    n[12] = 0x39;
    n[13] = 0x52;
    n[14] = 0x55;
    n[15] = 0x28;
    n[16] = 0xed;
    n[17] = 0x3f;
    n[18] = 0xa4;
    n[19] = 0x61;
    n[20] = 0xd3;
    n[21] = 0xf4;
    n[22] = 0xf2;
    n[23] = 0x34;
    n[24] = 0x6a;
    n[25] = 0x54;
    n[26] = 0xd1;
    n[27] = 0x15;
    n[28] = 0x7d;
    n[29] = 0x67;
    n[30] = 0xb;
    n[31] = 0xc7;
    n[32] = 0x8c;
    n[33] = 0xfe;
    n[34] = 0x1b;
    n[35] = 0x68;
    n[36] = 0x44;
    n[37] = 0x7;
    n[38] = 0x26;
    n[39] = 0x99;
    n[40] = 0xb;
    n[41] = 0x4d;
    n[42] = 0xc7;
    n[43] = 0x3f;
    n[44] = 0x52;
    n[45] = 0x90;
    n[46] = 0x2;
    n[47] = 0x68;
    n[48] = 0x3d;
    n[49] = 0x83;
    n[50] = 0x1d;
    n[51] = 0x79;
    n[52] = 0x7a;
    n[53] = 0x3f;
    n[54] = 0x36;
    n[55] = 0xf3;
    n[56] = 0x41;
    n[57] = 0x8b;
    n[58] = 0x7c;
    n[59] = 0xdf;
    n[60] = 0x64;
    n[61] = 0xac;
    n[62] = 0x74;
    n[63] = 0x7c;
    n[64] = 0x8;
    n[65] = 0xdb;
    n[66] = 0xa0;
    n[67] = 0x6f;
    n[68] = 0x10;
    n[69] = 0x71;
    n[70] = 0x13;
    n[71] = 0x86;
    n[72] = 0xaf;
    n[73] = 0xb8;
    n[74] = 0x71;
    n[75] = 0xf8;
    n[76] = 0xf0;
    n[77] = 0x45;
    n[78] = 0xa7;
    n[79] = 0x94;
    n[80] = 0xb3;
    n[81] = 0x6b;
    n[82] = 0x1e;
    n[83] = 0xff;
    n[84] = 0x8e;
    n[85] = 0x13;
    n[86] = 0xae;
    n[87] = 0xc2;
    n[88] = 0x59;
    n[89] = 0x56;
    n[90] = 0xd3;
    n[91] = 0xd;
    n[92] = 0x20;
    n[93] = 0x62;
    n[94] = 0x21;
    n[95] = 0x30;
    n[96] = 0x1d;
    n[97] = 0x6b;
    n[98] = 0x5e;
    n[99] = 0xc;
    n[100] = 0x0;
    n[101] = 0x35;
    n[102] = 0xae;
    n[103] = 0xbd;
    n[104] = 0xa5;
    n[105] = 0xc2;
    n[106] = 0x25;
    n[107] = 0x98;
    n[108] = 0xe7;
    n[109] = 0x57;
    n[110] = 0x89;
    n[111] = 0xc;
    n[112] = 0x12;
    n[113] = 0xf9;
    n[114] = 0x33;
    n[115] = 0x3d;
    n[116] = 0xa;
    n[117] = 0xac;
    n[118] = 0x51;
    n[119] = 0xd8;
    n[120] = 0x5c;
    n[121] = 0x40;
    n[122] = 0x9b;
    n[123] = 0xfa;
    n[124] = 0xf9;
    n[125] = 0xbc;
    n[126] = 0x3;
    n[127] = 0xe6;

    
	unsigned char *gpu_n;
	hipMalloc(&gpu_n, (4*n_size + n_size));
	hipMemcpy(gpu_n, n, (4*n_size + n_size), hipMemcpyHostToDevice);
//gpu_n和n是一样的
unsigned char *r = (unsigned char *) calloc(2*n_size, sizeof(char));
//
    r[0] = 0x7f;
    r[1] = 0x9d;
    r[2] = 0xe9;
    r[3] = 0x40;
    r[4] = 0x57;
    r[5] = 0x2;
    r[6] = 0x6e;
    r[7] = 0x93;
    r[8] = 0x2b;
    r[9] = 0xb4;
    r[10] = 0xe3;
    r[11] = 0xfd;
    r[12] = 0xba;
    r[13] = 0xc;
    r[14] = 0xcd;
    r[15] = 0x78;
    r[16] = 0x7d;
    r[17] = 0xae;
    r[18] = 0x8d;
    r[19] = 0x80;
    r[20] = 0xff;
    r[21] = 0x66;
    r[22] = 0x33;
    r[23] = 0xb;
    r[24] = 0x28;
    r[25] = 0x4c;
    r[26] = 0x93;
    r[27] = 0x30;
    r[28] = 0x2;
    r[29] = 0x92;
    r[30] = 0xa0;
    r[31] = 0x7c;
    r[32] = 0xf1;
    r[33] = 0xc;
    r[34] = 0xa;
    r[35] = 0x5e;
    r[36] = 0xf2;
    r[37] = 0x9a;
    r[38] = 0x8f;
    r[39] = 0x17;
    r[40] = 0x4c;
    r[41] = 0x82;
    r[42] = 0x25;
    r[43] = 0xe5;
    r[44] = 0x98;
    r[45] = 0x45;
    r[46] = 0x4d;
    r[47] = 0xc7;
    r[48] = 0xd9;
    r[49] = 0x53;
    r[50] = 0x5e;
    r[51] = 0x5a;
    r[52] = 0x6e;
    r[53] = 0x37;
    r[54] = 0x43;
    r[55] = 0x29;
    r[56] = 0x88;
    r[57] = 0xcb;
    r[58] = 0xe9;
    r[59] = 0x31;
    r[60] = 0x2f;
    r[61] = 0xd7;
    r[62] = 0x6;
    r[63] = 0xfb;
    r[64] = 0xf1;
    r[65] = 0x38;
    r[66] = 0xdf;
    r[67] = 0xc4;
    r[68] = 0xda;
    r[69] = 0x7c;
    r[70] = 0x9;
    r[71] = 0x5c;
    r[72] = 0xf9;
    r[73] = 0x2b;
    r[74] = 0x81;
    r[75] = 0x30;
    r[76] = 0xe9;
    r[77] = 0x29;
    r[78] = 0xcd;
    r[79] = 0x45;
    r[80] = 0xee;
    r[81] = 0xff;
    r[82] = 0x5b;
    r[83] = 0x3c;
    r[84] = 0x23;
    r[85] = 0x6d;
    r[86] = 0xb9;
    r[87] = 0xa1;
    r[88] = 0x89;
    r[89] = 0x3f;
    r[90] = 0xc3;
    r[91] = 0x9e;
    r[92] = 0xa1;
    r[93] = 0x30;
    r[94] = 0x98;
    r[95] = 0xf8;
    r[96] = 0xc8;
    r[97] = 0x4a;
    r[98] = 0xbe;
    r[99] = 0xc6;
    r[100] = 0x49;
    r[101] = 0xf7;
    r[102] = 0xb3;
    r[103] = 0xff;
    r[104] = 0x9;
    r[105] = 0x3b;
    r[106] = 0x94;
    r[107] = 0x9d;
    r[108] = 0x2f;
    r[109] = 0x5c;
    r[110] = 0x68;
    r[111] = 0xe1;
    r[112] = 0x6;
    r[113] = 0xf1;
    r[114] = 0x33;
    r[115] = 0xeb;
    r[116] = 0xc5;
    r[117] = 0x88;
    r[118] = 0xa5;
    r[119] = 0x1c;
    r[120] = 0xde;
    r[121] = 0x2c;
    r[122] = 0x64;
    r[123] = 0xad;
    r[124] = 0x5c;
    r[125] = 0xc9;
    r[126] = 0xeb;
    r[127] = 0x1c;
    r[128] = 0x1;

	unsigned char *gpu_r;
	hipMalloc(&gpu_r, (2*n_size));
	hipMemcpy(gpu_r, r, 2*n_size, hipMemcpyHostToDevice);

	unsigned char *message = (unsigned char *) calloc(n_size, sizeof(char));
	message[0] = 0x68;//h
	message[1] = 0x65;//e
	message[2] = 0x6c;//l
	message[3] = 0x6c;//l
	message[4] = 0x6f;//o


	unsigned char *gpu_message;
	hipMalloc(&gpu_message, n_size);
	hipMemcpy(gpu_message, message, n_size, hipMemcpyHostToDevice);

	unsigned char *exponent = (unsigned char *) malloc(3);
	exponent[0] = 0x01;
	exponent[1] = 0x00;
	exponent[2] = 0x01;
	unsigned int exponent_size = 3;

	//exponentiate m^e mod n
	//parameters: 
	//message(m)
	//exponent(e)
	//precomputation of r = floor((4^k)/n) where k is found by where (2^k) > n
	//modulus (n)

	unsigned char *cpu_ciphertext = (unsigned char *) calloc(n_size, sizeof(char));
	unsigned char *gpu_ciphertext;
	hipMalloc(&gpu_ciphertext, n_size);
	hipMemset(gpu_ciphertext, 0x00, n_size);

	unsigned char *m0_copy;
	hipMalloc(&m0_copy, n_size);

	unsigned char *reduction = (unsigned char *) calloc(n_size, sizeof(char));

	unsigned char *buf_cpu = (unsigned char *) calloc((n_size * 2) + 1, sizeof(char));
	unsigned char *buf_gpu;
	hipMalloc(&buf_gpu, ((n_size * 2) + 1));
	hipMemset(buf_gpu, 0x00, (n_size * 2) + 1);

	unsigned char *temp_cpu = (unsigned char *) calloc(3*n_size, sizeof(char));
	unsigned char *temp_gpu;
	hipMalloc(&temp_gpu, (3*n_size));
	
	unsigned char *shifted_cpu = (unsigned char *) calloc(n_size, sizeof(char));
	unsigned char *shifted_gpu;
	hipMalloc(&shifted_gpu, n_size);
	hipMemset(shifted_gpu, 0x00, n_size);

	unsigned char *xprime_cpu = (unsigned char *) calloc(2*n_size, sizeof(char));
	unsigned char *xprime_gpu;
	hipMalloc(&xprime_gpu, (2*n_size));
	hipMemset(xprime_gpu, 0x00, 2*n_size);

	unsigned char *result = (unsigned char *) calloc(n_size + 1, sizeof(char));
	
	unsigned char *tmp = (unsigned char *) calloc(n_size + 1, sizeof(char));

	unsigned int *transfer = (unsigned int *) calloc(4*n_size, sizeof(int));
	unsigned int *kernel_buf;
	hipMalloc(&kernel_buf, 4*n_size*sizeof(int));
	hipMemset(kernel_buf, 0x00, 4*n_size*sizeof(int));

	hipError_t error;
	hipEvent_t start;

	error = hipEventCreate(&start);
	if(error != hipSuccess)
	printf("error\n");

	hipEvent_t stop;
	error = hipEventCreate(&stop);
	if(error != hipSuccess)
	printf("error\n");

	error = hipEventRecord(start, NULL);


	exponentiation(gpu_message, exponent, gpu_ciphertext, m0_copy, reduction, buf_cpu, buf_gpu, temp_cpu, temp_gpu, shifted_cpu, shifted_gpu, xprime_cpu, xprime_gpu, result, tmp, exponent_size, gpu_r, gpu_n, kernel_buf, transfer, n);

	unsigned int d_exponent_size = 128;
	unsigned char *d_exponent = (unsigned char *) malloc(128);
    
    d_exponent[0] = 0x91;
    d_exponent[1] = 0xa;
    d_exponent[2] = 0xb3;
    d_exponent[3] = 0x66;
    d_exponent[4] = 0xbd;
    d_exponent[5] = 0x6f;
    d_exponent[6] = 0x18;
    d_exponent[7] = 0xde;
    d_exponent[8] = 0xd5;
    d_exponent[9] = 0x1;
    d_exponent[10] = 0x61;
    d_exponent[11] = 0x36;
    d_exponent[12] = 0x95;
    d_exponent[13] = 0x6d;
    d_exponent[14] = 0xdd;
    d_exponent[15] = 0x33;
    d_exponent[16] = 0xdb;
    d_exponent[17] = 0x26;
    d_exponent[18] = 0x3;
    d_exponent[19] = 0xe;
    d_exponent[20] = 0x68;
    d_exponent[21] = 0x54;
    d_exponent[22] = 0x73;
    d_exponent[23] = 0xa0;
    d_exponent[24] = 0xe0;
    d_exponent[25] = 0x6e;
    d_exponent[26] = 0x70;
    d_exponent[27] = 0x74;
    d_exponent[28] = 0x25;
    d_exponent[29] = 0x8b;
    d_exponent[30] = 0x2b;
    d_exponent[31] = 0xfb;
    d_exponent[32] = 0x9e;
    d_exponent[33] = 0x3c;
    d_exponent[34] = 0x34;
    d_exponent[35] = 0x2e;
    d_exponent[36] = 0x45;
    d_exponent[37] = 0x10;
    d_exponent[38] = 0x10;
    d_exponent[39] = 0x6c;
    d_exponent[40] = 0xfb;
    d_exponent[41] = 0xb7;
    d_exponent[42] = 0x9b;
    d_exponent[43] = 0xc8;
    d_exponent[44] = 0xcf;
    d_exponent[45] = 0x71;
    d_exponent[46] = 0xd9;
    d_exponent[47] = 0x96;
    d_exponent[48] = 0xb7;
    d_exponent[49] = 0xbb;
    d_exponent[50] = 0x5f;
    d_exponent[51] = 0x19;
    d_exponent[52] = 0x76;
    d_exponent[53] = 0x36;
    d_exponent[54] = 0x49;
    d_exponent[55] = 0x6a;
    d_exponent[56] = 0xb3;
    d_exponent[57] = 0x83;
    d_exponent[58] = 0xc3;
    d_exponent[59] = 0x59;
    d_exponent[60] = 0x2e;
    d_exponent[61] = 0x62;
    d_exponent[62] = 0x87;
    d_exponent[63] = 0xa2;
    d_exponent[64] = 0x5a;
    d_exponent[65] = 0x2f;
    d_exponent[66] = 0x60;
    d_exponent[67] = 0x75;
    d_exponent[68] = 0x1;
    d_exponent[69] = 0xf0;
    d_exponent[70] = 0x3f;
    d_exponent[71] = 0xdb;
    d_exponent[72] = 0x5a;
    d_exponent[73] = 0x70;
    d_exponent[74] = 0x1f;
    d_exponent[75] = 0x44;
    d_exponent[76] = 0x6a;
    d_exponent[77] = 0x9c;
    d_exponent[78] = 0x77;
    d_exponent[79] = 0x63;
    d_exponent[80] = 0xba;
    d_exponent[81] = 0xcb;
    d_exponent[82] = 0xcd;
    d_exponent[83] = 0x1f;
    d_exponent[84] = 0x99;
    d_exponent[85] = 0x70;
    d_exponent[86] = 0x89;
    d_exponent[87] = 0x94;
    d_exponent[88] = 0x31;
    d_exponent[89] = 0x2;
    d_exponent[90] = 0xa;
    d_exponent[91] = 0x32;
    d_exponent[92] = 0x96;
    d_exponent[93] = 0x65;
    d_exponent[94] = 0x21;
    d_exponent[95] = 0x21;
    d_exponent[96] = 0x59;
    d_exponent[97] = 0x55;
    d_exponent[98] = 0x8a;
    d_exponent[99] = 0xd0;
    d_exponent[100] = 0x7a;
    d_exponent[101] = 0x1c;
    d_exponent[102] = 0xd2;
    d_exponent[103] = 0x66;
    d_exponent[104] = 0x48;
    d_exponent[105] = 0x95;
    d_exponent[106] = 0x8;
    d_exponent[107] = 0xd3;
    d_exponent[108] = 0x6b;
    d_exponent[109] = 0xe7;
    d_exponent[110] = 0x9c;
    d_exponent[111] = 0xb9;
    d_exponent[112] = 0x96;
    d_exponent[113] = 0x20;
    d_exponent[114] = 0x20;
    d_exponent[115] = 0x8a;
    d_exponent[116] = 0xe5;
    d_exponent[117] = 0x4d;
    d_exponent[118] = 0x3e;
    d_exponent[119] = 0x53;
    d_exponent[120] = 0x4b;
    d_exponent[121] = 0xd8;
    d_exponent[122] = 0x21;
    d_exponent[123] = 0x4;
    d_exponent[124] = 0x81;
    d_exponent[125] = 0x7d;
    d_exponent[126] = 0x29;
    d_exponent[127] = 0x38;


	memset(message, 0x00, n_size);
	hipMemset(gpu_message, 0x00, n_size);

	exponentiation(gpu_ciphertext, d_exponent, gpu_message, m0_copy, reduction, buf_cpu, buf_gpu, temp_cpu, temp_gpu, shifted_cpu, shifted_gpu, xprime_cpu, xprime_gpu, result, tmp, d_exponent_size, gpu_r, gpu_n, kernel_buf, transfer, n);


	error = hipEventRecord(stop, NULL);

	error = hipEventSynchronize(stop);

	if(error != hipSuccess)
		printf("error\n");

	float msecTotal = 0.0f;
	error = hipEventElapsedTime(&msecTotal, start, stop);
	printf("GPU time: %.6f\n", msecTotal / 1000);


	hipMemcpy(message, gpu_message, n_size, hipMemcpyDeviceToHost);
	int z = 0;
	while (z < n_size) {
		printf("message[%d] = %x\n", z, message[z]);
		z++;
	}

	return 0;
}

void exponentiation(unsigned char *gpu_message, unsigned char *exponent, unsigned char *gpu_ciphertext, unsigned char *m0_copy, unsigned char *reduction, unsigned char *buf_cpu, unsigned char *buf_gpu, unsigned char *temp_cpu, unsigned char *temp_gpu, unsigned char *shifted_cpu, unsigned char *shifted_gpu, unsigned char *xprime_cpu, unsigned char *xprime_gpu, unsigned char *result, unsigned char *tmp, unsigned int exponent_size, unsigned char *gpu_r, unsigned char *gpu_n, unsigned int *kernel_buf, unsigned int *transfer, unsigned char *n) {


	dim3 blocksPerGrid(2);
	dim3 threadsPerBlock(64);


	//get the total amount of bits in strlen(exponent) zero based
	//not including the final char index msb (byte)
	unsigned int total_bits = exponent_size * 8 - 1;
	
	//find the most signinficant bit in the most significant byte (char index)
	//find most significant bit in exponent[exp_size - 1]
	unsigned char mask = 0x80; //10000000 in binary
	unsigned char msb = 0;
	int i = 0;
	while(i < 8) {
		if((exponent[exponent_size - 1] & (mask >> i)) == (mask >> i)) {
			msb = i;
			break;
		}
		i++;
	}

	//subtract most significant bit from total_bits to know total amount of significant bits
	//for loop of exponent in binary
	unsigned int exp_bits = (total_bits - msb);

	//keep copy of original message m0
	hipMemcpy(m0_copy, gpu_message, n_size, hipMemcpyDeviceToDevice);

	//compute m^e where e is in binary 
	//RULES:
		//iterate over the values of msb to 0 bit by bit
		//msb is amount of relevent bits to check for exponentiation
		//total bits is the amount of total bits in exponent lenth
		//square m(current) for each itteration
		//check if current bit is 1
		//current bit is 1: m(current) * m0
		//curent bit is 0: return to loop


	//subtract one from total because to exponentiate in binary
	//start at the second bit after the most significant bit
	//each bit equals m^2 and when the current bit is 1 it is
	//(m^2)*m0 or if it is 0 then m^2
	int index_of_bit = exp_bits - 1; //subtraction of 1 is becuase msb is zero based
	while (index_of_bit >= 0) {

		//allocate space for reduction to hold a value strickly less than n
		//buf holds value at most m^2 which is less than n^2

		//calculate m^2
		get_square<<<blocksPerGrid, threadsPerBlock>>>(gpu_message, kernel_buf, n_size);
		hipMemcpy(transfer, kernel_buf, 2*n_size*sizeof(unsigned int), hipMemcpyDeviceToHost);

		unsigned int index = 0;
		while(index < 2*n_size) {
			buf_cpu[index] = (unsigned char) transfer[index];
			transfer[index + 1] += (unsigned int) (transfer[index]>>8);
			index++;
		}
	
		memset(transfer, 0x00, 4*n_size*sizeof(int));	
		hipMemset(kernel_buf, 0x00, 4*n_size*sizeof(int));
		hipMemcpy(buf_gpu, buf_cpu, 2*n_size, hipMemcpyHostToDevice);

		//calculate m^2 mod n
		barrett_reduction(buf_cpu, buf_gpu, gpu_r, gpu_n, reduction, temp_cpu, temp_gpu, shifted_cpu, shifted_gpu, xprime_cpu, xprime_gpu, result, tmp, exponent_size, kernel_buf, transfer, n);

		hipMemcpy(gpu_message, reduction, n_size, hipMemcpyHostToDevice);

		hipMemset(buf_gpu, 0x00, 2*n_size);
		memset(buf_cpu, 0x00, 2*n_size);
		memset(reduction, 0x00, n_size);

		char bit;
		if ((bit = checkbit(exponent, index_of_bit)) == 1) {
			//m * m0
			get_products<<<blocksPerGrid, threadsPerBlock>>>(gpu_message, m0_copy, kernel_buf, n_size);
			hipMemcpy(transfer, kernel_buf, 2*n_size*sizeof(unsigned int), hipMemcpyDeviceToHost);

			index = 0;
			while(index < 2*n_size) {
				buf_cpu[index] = (unsigned char) transfer[index];
				transfer[index + 1] += (unsigned int) (transfer[index]>>8);
				index++;
			}
		
			memset(transfer, 0x00, 4*n_size*sizeof(int));
			hipMemset(kernel_buf, 0x00, 4*n_size*sizeof(int));
			hipMemcpy(buf_gpu, buf_cpu, 2*n_size, hipMemcpyHostToDevice);



			barrett_reduction(buf_cpu, buf_gpu, gpu_r, gpu_n, reduction, temp_cpu, temp_gpu, shifted_cpu, shifted_gpu, xprime_cpu, xprime_gpu, result, tmp, exponent_size, kernel_buf, transfer, n);

			hipMemcpy(gpu_message, reduction, n_size, hipMemcpyHostToDevice);
			hipMemset(buf_gpu, 0x00, 2*n_size);
			memset(buf_cpu, 0x00, 2*n_size);
			memset(reduction, 0x00, n_size);
		
		}

		index_of_bit--;
	}
	
	//copy back final value of message to ciphertext for decryption
	hipMemcpy(gpu_ciphertext, gpu_message, n_size, hipMemcpyDeviceToHost);

	hipMemset(m0_copy, 0x00, n_size);

	return;
}
void barrett_reduction(unsigned char *buf_cpu, unsigned char *buf_gpu, unsigned char *gpu_r, unsigned char *gpu_n, unsigned char *reduction, unsigned char *temp_cpu, unsigned char *temp_gpu, unsigned char *shifted_cpu, unsigned char *shifted_gpu, unsigned char *xprime_cpu, unsigned char *xprime_gpu, unsigned char *result, unsigned char *tmp, unsigned int exp_size, unsigned int *kernel_buf, unsigned int *transfer, unsigned char *n) {
	////////////////////////////////////////////////////////////////////////////////////

	////////////////////  calculate: t = x - ((x*r)/(4^k))*n  /////////////////////////

	///////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////
	//multiply: x * r = temp
	//size of x is assumed to be the largest value which is = largest value of 2*n
	//size of r is precomputed
	dim3 blocksPerGrid_two(4);
	dim3 threadsPerBlock_two(64);

	get_products<<<blocksPerGrid_two, threadsPerBlock_two>>>(gpu_r, buf_gpu, kernel_buf, 2*n_size);
	hipMemcpy(transfer, kernel_buf, 3*n_size*sizeof(unsigned int), hipMemcpyDeviceToHost);

	unsigned int index = 0;
	while(index < 3*n_size) {
		temp_cpu[index] = (unsigned char) transfer[index];
		transfer[index + 1] += (unsigned int) (transfer[index]>>8);
		index++;
	}

	hipMemset(kernel_buf, 0x00, 4*n_size*sizeof(int));

	//shift bits by (4^k) or (2^(2*k))
	//shift temp by 2*k store to shifted
	//size of shifted is 2*n + sizeof(r)
	
	//find the actual amount of bits/bytes left in the value of temp
	//which is equal to x * r so that the correct size of the value
	//can be used in the bit_shift function

	//first find the amount of bytes from most significant byte
	//to least and then when one char does not equal to 0x00

	unsigned int zero_bytes = 0;
	int count = (3*n_size) - 1;
	while((count >= 0) && (temp_cpu[count] == 0x00)) {
		count--;
		zero_bytes++;
	}


	bit_shift(temp_cpu, shifted_cpu, k, (3*n_size) - zero_bytes);

	//multiply: shifted * n = xprime
	//xprime is the size of 2*n + R_size - (k >> 0x07) + n
	hipMemcpy(shifted_gpu, shifted_cpu, n_size, hipMemcpyHostToDevice);

	dim3 blocksPerGrid_one(2);
	dim3 threadsPerBlock_one(64);
	get_products<<<blocksPerGrid_one, threadsPerBlock_one>>>(shifted_gpu, gpu_n, kernel_buf, n_size);
	hipMemcpy(transfer, kernel_buf, 2*n_size*sizeof(unsigned int), hipMemcpyDeviceToHost);

	index = 0;
	while(index < 2*n_size) {
		xprime_cpu[index] = (unsigned char) transfer[index];
		transfer[index + 1] += (unsigned int) (transfer[index]>>8);
		index++;
	}



	//subtract xprime from x^2
	hipMemset(kernel_buf, 0x00, 4*n_size*sizeof(int));
	subtraction(buf_cpu, xprime_cpu, result, 2*n_size);

	//the field of n, if the value is not within the field of n then reduce the value by subtracting
	//the value of result = t - n which is guaranteed to be in the field of n
	if ((result[n_size] == 0x00) && (result[n_size - 1] < n[n_size - 1])) {
		memcpy(reduction, result, n_size);
	}

	else {
		unsigned char *tmp = (unsigned char *) calloc(n_size + 1, sizeof(char));
		subtraction(result, n, tmp, n_size + 1);
		memcpy(reduction, tmp, n_size);
		memset(tmp, 0x00, n_size + 1);
	}


	memset(temp_cpu, 0x00, 3*n_size);
	hipMemset(temp_gpu, 0x00, 3*n_size);
	memset(shifted_cpu, 0x00, n_size);
	hipMemset(shifted_gpu, 0x00, n_size);
	memset(xprime_cpu, 0x00, 2*n_size);
	hipMemset(xprime_gpu, 0x00, 2*n_size);
	memset(result, 0x00, n_size + 1);
	memset(transfer, 0x00, 4*n_size*sizeof(int));


	return;
}

void subtraction(unsigned char *a, unsigned char *b, unsigned char *c, unsigned int size) {

	//borrow represents the value 1 or 0 for the current index
	//indecating if the current index has been borrowed from by
	//the previous index, borrow = 1 true, 0 false
	unsigned char borrow = 0x00; //value is 0 or 1

	//loop through array a size and subtract a - b,
	//a is guaranted to be greater than b in 
	//barrett reduction
	unsigned int i = 0;
	while(i < size) {
		//check current value of a to make sure that it is
		//not 0 when the previous index has borrowed
		if (a[i] == 0 && borrow == 1) {
			//borrow from next sequential index with
			//0x100 and subtract 0x01 for the
			//previous borrow which is = 0xff
			c[i] = 0xff - b[i];
			//turn on borrow for next index
			borrow = 0x01;
			i++;
			continue;
		}

		//calculate current value of a along with if the 
		//previous index has borrowed
		a[i] = a[i] - borrow;
	
		//calculate the value of a - b only when a - b >= 0
		//borrow has already been accounted for
		if (a[i] >= b[i]) {
			c[i] = a[i] - b[i];
			borrow = 0x00;
		}

		//a - b !> 0, borrow from next sequential index by 
		//taking the value 0x100 and adding to a[i] and 
		//subtracting b[i] which will give a value between
		//{0x01...0xff} and turn on borrow for next index
		else {
			c[i] = 0x100 + a[i] - b[i];
			borrow = 0x01;
		}
		i++;
	}
	return;
}

char checkbit(unsigned char *exponent, unsigned int index_of_bit) {

	unsigned char bit;
	//get the characters index of which the bit is located in by 
	//taking index_of_bit which is the size of the bits left to 
	//check and divide by 8 giving the location index of the
	//current bit to be checked
	unsigned int quotient = (index_of_bit >> 0x03); // index_of_bit / 8

	//find the bit within the index previously found by finding the 
	//remainder of 8 % index of bit, this will locate the exact
	//bit to be checked
	unsigned int remainder = index_of_bit & (0x07); // index_of_bit % 8
	
	//mask is equivelent to 1 in order to compare a single bit with a
	//the current bit to be checked
	unsigned char mask = 0x01; // use single bit to mask with selected bit
	
	//use the remainder by knowing the index of the character and 
	//the remainder allows the bit to be shifted to the position of
	//the current bit to be checked
	mask = mask << remainder; // shift single bit to bit_in_index position
	
	//bit is now located at index_of_bit character index of array
	//and bit location bit_in_index in group of 8 bits at index
	bit = (exponent[quotient] & mask); // & to see if single bit is on or off

	//shift bit back to the 1 position to represent value 1 or 0
	bit = bit >> remainder; //shift bit back to value of one or zero
	
	return bit;
}

//b is expected to be completely zero before shift
void bit_shift(unsigned char *a, unsigned char *b, unsigned int k_val, unsigned int size_of_a) {

        //expected that k will be equivlent to some power of 2
        //represents the division of (4^k) which is = (2^(2*k))
        unsigned int shift = k_val * 2;

        //quotient represents groups of 8 bits that equal 0 as in >> 8 in single char
        //leaving it to be the value of 0x00
        unsigned int quotient = shift >> 0x03; // k / 8 as integer

	//in case that the shift is greater than the actual value of the
	//number being shifted
	if(quotient > size_of_a) {
		return;
	}

        //printf("quotient = %d\n", quotient);
        //remainder will find final char index shift value = {0...7}
        //the specific bits to be shifted in the last group which is not greater than 7
        unsigned int remainder = shift & 0x07; // k % 8 
        //printf("remainder = %d\n", remainder);
        //move a to b by shifting the characters an index of quotient amount
        //and then use the remainder to shift the final index to correct 
        //position
        unsigned int constant = (size_of_a - quotient);

        unsigned int j = 0;
        while (j < constant) {
                b[j] = a[quotient + j] >> remainder;
                unsigned char cpy_bits = a[quotient + j + 1] << (8 - remainder);
                b[j] = b[j] | cpy_bits;
                j++;
        }

        return;
}

__global__ void get_products(unsigned char a[], unsigned char b[], unsigned int accumulator[], unsigned int n) {

        int multiplier = 0;
        unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
        unsigned int multiplicand = index;
        unsigned int product = 0;

        while(multiplier < n) {
                product = (unsigned int) a[multiplier] * b[multiplicand];
                atomicAdd(&accumulator[multiplier + index], product<<24>>24);
                atomicAdd(&accumulator[multiplier + index + 1], product>>8);
                multiplier++;
        }

	return;
}


__global__ void get_square(unsigned char a[], unsigned int accumulator[], unsigned int n) {

        int multiplier = 0;
        unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
        unsigned int multiplicand = index;
        unsigned int product = 0;

        while(multiplier < n) {
                product = (unsigned int) a[multiplier] * a[multiplicand];
                atomicAdd(&accumulator[multiplier + index], product<<24>>24);
                atomicAdd(&accumulator[multiplier + index + 1], product>>8);
                multiplier++;
        }
	return;
}
