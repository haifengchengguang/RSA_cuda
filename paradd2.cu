// parallel add of large integers
// requires CC 2.0 or higher
// compile with:
// nvcc -O3 -arch=sm_20 -o paradd2 paradd2.cu

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define MAXSIZE 1024 // the number of 64 bit quantities that can be added
#define LLBITS 64  // the number of bits in a long long
#define BSIZE ((MAXSIZE + LLBITS -1)/LLBITS) // MAXSIZE when packed into bits
#define nTPB MAXSIZE

// define either GPU or GPUCOPY, not both -- for timing
#define GPU
//#define GPUCOPY

#define LOOPCNT 1000

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

// perform c = a + b, for unsigned integers of psize*64 bits.
// all work done in a single threadblock.
// multiple threadblocks are handling multiple separate addition problems
// least significant word is at a[0], etc.

__global__ void paradd(const unsigned size, const unsigned psize, unsigned long long *c, const unsigned long long *a, const unsigned long long *b){

  __shared__ unsigned long long carry_through[BSIZE];
  __shared__ unsigned long long carry[BSIZE+1];
  __shared__ volatile unsigned mcarry;
  __shared__ volatile unsigned mcarry_through;

  unsigned idx = threadIdx.x + (psize * blockIdx.x);
  if ((threadIdx.x < psize) && (idx < size)){
    // handle 64 bit unsigned add first
    unsigned long long cr1 = a[idx];
    unsigned long long lc = cr1 + b[idx];
    // handle carry
    if (threadIdx.x < BSIZE){
      carry[threadIdx.x] = 0;
      carry_through[threadIdx.x] = 0;
      }
    if (threadIdx.x == 0){
      mcarry = 0;
      mcarry_through = 0;
      }
    __syncthreads();
    if (lc < cr1){
      if ((threadIdx.x%LLBITS) != (LLBITS-1))  
        atomicAdd(&(carry[threadIdx.x/LLBITS]), (2ull<<(threadIdx.x%LLBITS)));
      else atomicAdd(&(carry[(threadIdx.x/LLBITS)+1]), 1);
      }
    // handle carry-through
    if (lc == 0xFFFFFFFFFFFFFFFFull) 
      atomicAdd(&(carry_through[threadIdx.x/LLBITS]), (1ull<<(threadIdx.x%LLBITS))); 
    __syncthreads();
    if (threadIdx.x < ((psize + LLBITS-1)/LLBITS)){
      // only 1 warp executing within this if statement
      unsigned long long cr3 = carry_through[threadIdx.x];
      cr1 = carry[threadIdx.x] & cr3;
      // start of sub-add
      unsigned long long cr2 = cr3 + cr1;
      if (cr2 < cr1) atomicAdd((unsigned *)&mcarry, (2u<<(threadIdx.x)));
      if (cr2 == 0xFFFFFFFFFFFFFFFFull) atomicAdd((unsigned *)&mcarry_through, (1u<<threadIdx.x));
      if (threadIdx.x == 0) {
        unsigned cr4 = mcarry & mcarry_through;
        cr4 += mcarry_through;
        mcarry |= (mcarry_through ^ cr4); 
        }
      if (mcarry & (1u<<threadIdx.x)) cr2++;
      // end of sub-add
      carry[threadIdx.x] |= (cr2 ^ cr3);
      }
    __syncthreads();
    if (carry[threadIdx.x/LLBITS] & (1ull<<(threadIdx.x%LLBITS))) lc++;
    c[idx] = lc;
  }
}

int main() {

  unsigned long long *h_a, *h_b, *h_c, *d_a, *d_b, *d_c, *c;
  unsigned at_once = 256;   // valid range = 1 .. 65535
  unsigned prob_size = MAXSIZE ; // valid range = 1 .. MAXSIZE
  unsigned dsize = at_once * prob_size;
  hipEvent_t t_start_gpu, t_start_cpu, t_end_gpu, t_end_cpu;
  float et_gpu, et_cpu, tot_gpu, tot_cpu;
  tot_gpu = 0;
  tot_cpu = 0;


  if (sizeof(unsigned long long) != (LLBITS/8)) {printf("Word Size Error\n"); return 1;}
  if ((c = (unsigned long long *)malloc(dsize * sizeof(unsigned long long)))  == 0) {printf("Malloc Fail\n"); return 1;}

  hipHostAlloc((void **)&h_a, dsize * sizeof(unsigned long long), hipHostMallocDefault);
  cudaCheckErrors("cudaHostAlloc1 fail");
  hipHostAlloc((void **)&h_b, dsize * sizeof(unsigned long long), hipHostMallocDefault);
  cudaCheckErrors("cudaHostAlloc2 fail");
  hipHostAlloc((void **)&h_c, dsize * sizeof(unsigned long long), hipHostMallocDefault);
  cudaCheckErrors("cudaHostAlloc3 fail");

  hipMalloc((void **)&d_a, dsize * sizeof(unsigned long long));
  cudaCheckErrors("cudaMalloc1 fail");
  hipMalloc((void **)&d_b, dsize * sizeof(unsigned long long));
  cudaCheckErrors("cudaMalloc2 fail");
  hipMalloc((void **)&d_c, dsize * sizeof(unsigned long long));
  cudaCheckErrors("cudaMalloc3 fail");
  hipMemset(d_c, 0, dsize*sizeof(unsigned long long));

  hipEventCreate(&t_start_gpu);
  hipEventCreate(&t_end_gpu);
  hipEventCreate(&t_start_cpu);
  hipEventCreate(&t_end_cpu);

  for (unsigned loops = 0; loops <LOOPCNT; loops++){
  //create some test cases
  if (loops == 0){
  for (int j=0; j<at_once; j++)
  for (int k=0; k<prob_size; k++){
    int i= (j*prob_size) + k;
    h_a[i] = 0xFFFFFFFFFFFFFFFFull;
    h_b[i] = 0;
    }
    h_a[prob_size-1] = 0;
    h_b[prob_size-1] = 1;
    h_b[0] = 1;
  }
  else if (loops == 1){
  for (int i=0; i<dsize; i++){
    h_a[i] = 0xFFFFFFFFFFFFFFFFull;
    h_b[i] = 0;
    }
    h_b[0] = 1;
  }
  else if (loops == 2){
  for (int i=0; i<dsize; i++){
    h_a[i] = 0xFFFFFFFFFFFFFFFEull;
    h_b[i] = 2;
    }
    h_b[0] = 1;
  }
  else {
  for (int i = 0; i<dsize; i++){
    h_a[i] = (((unsigned long long)rand())<<33) + (unsigned long long)rand();
    h_b[i] = (((unsigned long long)rand())<<33) + (unsigned long long)rand();
    }
  }
#ifdef GPUCOPY
  cudaEventRecord(t_start_gpu, 0);
#endif
  hipMemcpy(d_a, h_a, dsize*sizeof(unsigned long long), hipMemcpyHostToDevice);
  cudaCheckErrors("cudaMemcpy1 fail");
  hipMemcpy(d_b, h_b, dsize*sizeof(unsigned long long), hipMemcpyHostToDevice);
  cudaCheckErrors("cudaMemcpy2 fail");
#ifdef GPU
  hipEventRecord(t_start_gpu, 0);
#endif
  paradd<<<at_once, nTPB>>>(dsize, prob_size, d_c, d_a, d_b);
  cudaCheckErrors("Kernel Fail");
#ifdef GPU
  hipEventRecord(t_end_gpu, 0);
#endif
  hipMemcpy(h_c, d_c, dsize*sizeof(unsigned long long), hipMemcpyDeviceToHost);
  cudaCheckErrors("cudaMemcpy3 fail");
#ifdef GPUCOPY
  cudaEventRecord(t_end_gpu, 0);
#endif
  hipEventSynchronize(t_end_gpu);
  hipEventElapsedTime(&et_gpu, t_start_gpu, t_end_gpu);
  tot_gpu += et_gpu;
  hipEventRecord(t_start_cpu, 0);
  //also compute result on CPU for comparison
  for (int j=0; j<at_once; j++) {
    unsigned rc=0;
    for (int n=0; n<prob_size; n++){
      unsigned i = (j*prob_size) + n;
      c[i] = h_a[i] + h_b[i];
      if (c[i] < h_a[i]) {
        c[i] += rc;
        rc=1;}
      else {
        if ((c[i] += rc) != 0) rc=0;
        }
      if (c[i] != h_c[i]) {printf("Results mismatch at offset %d, GPU = 0x%lX, CPU = 0x%lX\n", i, h_c[i], c[i]); return 1;}
      }
    }
  hipEventRecord(t_end_cpu, 0);
  hipEventSynchronize(t_end_cpu);
  hipEventElapsedTime(&et_cpu, t_start_cpu, t_end_cpu);
  tot_cpu += et_cpu;
  if ((loops%(LOOPCNT/10)) == 0) printf("*\n");
  }
  printf("\nResults Match!\n");
  printf("Average GPU time = %fms\n", (tot_gpu/LOOPCNT));
  printf("Average CPU time = %fms\n", (tot_cpu/LOOPCNT));

  return 0;
}